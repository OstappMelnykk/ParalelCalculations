#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <vector>
#include <chrono>
#include <iomanip>
#include <algorithm>
#include <cassert>

using std::cout;
using std::vector;

__global__ void matrixMul(const int* a, const int* b, int* c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int tmp = 0;
        for (int k = 0; k < N; k++)
            tmp += a[row * N + k] * b[k * N + col];
        c[row * N + col] = tmp;
    }
}

void verify_result(const vector<int>& a, const vector<int>& b, const vector<int>& c, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int tmp = 0;
            for (int k = 0; k < N; k++)
                tmp += a[i * N + k] * b[k * N + j];

            if (tmp != c[i * N + j]) {
                cout << "Mismatch at position [" << i << "][" << j << "]: CPU result = " << tmp << ", GPU result = " << c[i * N + j] << std::endl;
                return;
            }
        }
    }
    cout << "Results verified: CPU and GPU results match." << std::endl;
}

int main() {

    cout << "Start: " << std::endl;

    int N = 2000;
    size_t bytes = N * N * sizeof(int);

    vector<int> h_a(N * N);
    vector<int> h_b(N * N);
    vector<int> h_c(N * N);

    std::generate(h_a.begin(), h_a.end(), []() { return rand() % 112; });
    std::generate(h_b.begin(), h_b.end(), []() { return rand() % 112; });

    int* d_a, * d_b, * d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);


    dim3 threads(32, 32);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);

    auto startTime = std::chrono::high_resolution_clock::now();

    matrixMul << <blocks, threads >> > (d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    auto endTime = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();

    hipError_t cudaKernelStatus = hipGetLastError();
    if (cudaKernelStatus != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(cudaKernelStatus));
    }

    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    cout << "Results verification: " << std::endl;
    //verify_result(h_a, h_b, h_c, N);

    std::cout << "Array size: " << N << "*" << N << "\nthreads:\n\tX:" << threads.x << "\n\tY:" << threads.y << "\nblocks:\n\t" << blocks.x << "\n\t" << blocks.y << "\nElapsed Time: " << duration << " ms" << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}